#include "hip/hip_runtime.h"
/**
Authors: Christian Henn, Qianli Liao
**/

#include <torch/types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cmath>
#include <cstdio>
#include <cstdint>
#include <cassert>
#include <iostream>

// define for error checking
//#define CUDA_ERROR_CHECK

#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    do{
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }

        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while(0);
#endif

    return;
}




const int BUFF_W = 1920;
const int BUFF_H = 1080;
const int BUFF_D = 3;

const int Z_MAX = 268435455;


__global__ void render_kernel(
        const u_int8_t** img_ptrs,
        const int* img_sizes,
        const int* depths,
        const int* locs_abs,

        int* z_buffer,
        int* f_buffer,
        int* lock_buffer
){

    // each block takes one image at img_ptr_i - of size img_size0 x img_size1 x 3
    const auto img_i = blockIdx.x;

    const auto img_ptr_i = img_ptrs[img_i];

    const auto img_size_0 = img_sizes[img_i * 2 + 0];
    const auto img_size_1 = img_sizes[img_i * 2 + 1];

    // convert to per-pixel depth when rendering 3D
    const auto im_depth = depths[img_i];

    const auto im_start_y = locs_abs[img_i * 2 + 0];
    const auto im_start_x = locs_abs[img_i * 2 + 1];


    // loop over pixels in block's image at img_ptr in x and y, for all 3 channels
    for (int im_row = threadIdx.x; im_row < img_size_0; im_row += blockDim.x){

        auto buff_y = im_start_y + im_row;
        if ( buff_y < 0 ) continue;
        if ( buff_y >= BUFF_H ) break;

        for (int im_col = 0; im_col < img_size_1; im_col++){

            auto buff_x = im_start_x + im_col;
            if ( buff_x < 0 ) continue;
            if ( buff_x >= BUFF_W ) break;

            // If my thread's z-depth is < the current value at my loc in z_buff: update z_buff, write my pixel value to f_buff
            // z-buff comparison and write must be atomic on this pixel in buffs
            bool holding_lock;
            do {
                holding_lock = (atomicCAS(&lock_buffer[buff_y * BUFF_W + buff_x], 0, -1) == 0);

                if (holding_lock)
                {
                    // z_buff is H x W
                    auto curr_z = z_buffer[buff_y * BUFF_W + buff_x];

                    if (im_depth < curr_z) {

                        z_buffer[buff_y * BUFF_W + buff_x] = im_depth;

                        // write pixel at this buff_loc to f_buff
                        // #pragma unroll
                        for (int chan = 0; chan < BUFF_D; chan++) {

                            auto lin_im_loc = (im_row * img_size_1 * BUFF_D) + (im_col * BUFF_D) + chan;
                            auto val = img_ptr_i[lin_im_loc];

                            // f_buff is H x W x D
                            auto lin_buff_loc = (buff_y * BUFF_W * BUFF_D) + (buff_x * BUFF_D) + chan;
                            f_buffer[lin_buff_loc] = val;
                        }
                    }
                    atomicExch(&lock_buffer[buff_y * BUFF_W + buff_x], 0);
                }

            } while (!holding_lock);
        }
    }

}

__global__ void transform_locs(
        const u_int8_t* child_mat,
        const int child_mat_size0,
        const int child_mat_size1,

        const int* locs_rel,
        int* locs_abs

){
    // locs_rel and locs_abs have width == 2

    // for each parent with id == row. cannot be visited in parallel.
    for (int row = 0; row < child_mat_size0; row++){
        auto curr_offs_0 = locs_abs[row * 2 + 0];
        auto curr_offs_1 = locs_abs[row * 2 + 1];

        // col gives id of child of curr object, if child_mat[row][col]==1. Can be visited in parallel
        for (int col = threadIdx.x; col < child_mat_size1; col += blockDim.x){

            if (!child_mat[row * child_mat_size1 + col]) continue;

            // abs loc of child = child_rel_offs + parent_abs_loc
            auto target_rel_0 = locs_rel[col * 2 + 0];
            auto target_rel_1 = locs_rel[col * 2 + 1];

            target_rel_0 += curr_offs_0;
            target_rel_1 += curr_offs_1;

            locs_abs[col * 2 + 0] = target_rel_0;
            locs_abs[col * 2 + 1] = target_rel_1;
        }
    }
}



__host__ std::vector<torch::Tensor> render_call(
    std::vector<torch::Tensor> imgs,
    torch::Tensor img_sizes,
    torch::Tensor child_mat,
    torch::Tensor locs_rel,
    torch::Tensor locs_abs,
    torch::Tensor depths,

    torch::Tensor z_buffer,
    torch::Tensor f_buffer,
    torch::Tensor lock_buffer
) {

    using namespace torch::indexing;
    auto device_id = child_mat.get_device();
    hipSetDevice(device_id);

    // convert std::vector of imgs into an array of device pointers, to pass to device code
    u_int8_t** img_ptrs;
    hipMalloc((void**) &img_ptrs, imgs.size() * sizeof(u_int8_t*));

    u_int8_t** tmp_d_ptrs = (u_int8_t**) malloc(imgs.size() * sizeof(u_int8_t*));
    for (int i = 0; i < imgs.size(); i++){
        tmp_d_ptrs[i] = imgs[i].data_ptr<u_int8_t>();
    }
    hipMemcpy(img_ptrs, tmp_d_ptrs, imgs.size() * sizeof(u_int8_t*), hipMemcpyHostToDevice);

    // refresh persistent buffers
    z_buffer.fill_(Z_MAX);
    f_buffer.fill_(0);
    lock_buffer.fill_(0);

    // transform relative locs to abs locs within the f_buff
    auto threads = child_mat.size(1);
    transform_locs<<<1,threads>>>(
        child_mat.data_ptr<u_int8_t>(),
        child_mat.size(0),
        child_mat.size(1),

        locs_rel.data_ptr<int>(),
        locs_abs.data_ptr<int>()
    );

    // render imgs onto f_buff
    auto n_threads = 256;
    render_kernel<<<imgs.size(),n_threads>>>(
        img_ptrs,
        img_sizes.data_ptr<int>(),
        depths.data_ptr<int>(),
        locs_abs.data_ptr<int>(),

        z_buffer.data_ptr<int>(),
        f_buffer.data_ptr<int>(),
        lock_buffer.data_ptr<int>()
    );

    return {};
}



